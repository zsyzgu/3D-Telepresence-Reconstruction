#include "hip/hip_runtime.h"
#include "CudaHandleError.h"
#include "Parameters.h"
#include "DepthFilter.h"
#include "Timer.h"

namespace FilterNamespace {
	__constant__ int SF_RADIUS = 5;
	__constant__ float SF_ALPHA = 0.75f;
	__constant__ float SF_THRESHOLD = 40.0f;
	__constant__ float TF_ALPHA = 0.5f;
	__constant__ float TF_THRESHOLD = 40.0f;

	float* lastFrame;
};
using namespace FilterNamespace;

__global__ void kernelCleanLastFrame(float* lastFrame) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < DEPTH_W && y < DEPTH_H) {
		int id = y * DEPTH_W + x;
		lastFrame[id] = 0;
	}
}

__global__ void kernelFilterToDisparity(UINT16* source, float* target, float convertFactor) {
	#define DEPTH_SORT(a, b) { if ((a) > (b)) {UINT16 temp = (a); (a) = (b); (b) = temp;} }

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < DEPTH_W && y < DEPTH_H) {
		int id = y * DEPTH_W + x;
		UINT16 arr[5] = { source[id], 0, 0, 0, 0 };
		if (x - 1 >= 0) arr[1] = source[id - 1];
		if (x + 1 < DEPTH_W) arr[2] = source[id + 1];
		if (y - 1 >= 0) arr[3] = source[id - DEPTH_W];
		if (y + 1 < DEPTH_H) arr[4] = source[id + DEPTH_W];
		DEPTH_SORT(arr[0], arr[1]);
		DEPTH_SORT(arr[0], arr[2]);
		DEPTH_SORT(arr[0], arr[3]);
		DEPTH_SORT(arr[0], arr[4]);
		DEPTH_SORT(arr[1], arr[2]);
		DEPTH_SORT(arr[1], arr[3]);
		DEPTH_SORT(arr[1], arr[4]);
		DEPTH_SORT(arr[2], arr[3]);
		DEPTH_SORT(arr[2], arr[4]);
		DEPTH_SORT(arr[3], arr[4]);
		__syncthreads();
		if (arr[0] != 0) {
			target[id] = convertFactor / arr[2];
		} else
		if (arr[1] != 0) {
			target[id] = convertFactor * 2 / (arr[2] + arr[3]);
		} else
		if (arr[2] != 0) {
			target[id] = convertFactor / arr[3];
		} else
		if (arr[3] != 0) {
			target[id] = convertFactor * 2 / (arr[3] + arr[4]);
		} 
		if (arr[4] != 0) {
			target[id] = convertFactor / arr[4];
		} else {
			target[id] = 0;
		}
	}
}

__global__ void kernelFilterToDepth(float* depth, float convertFactor) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < DEPTH_W && y < DEPTH_H) {
		int id = y * DEPTH_W + x;
		if (depth[id] != 0) {
			depth[id] = convertFactor / depth[id] * 0.001; //to m
		}
	}
}

__global__ void kernelSFVertical(float* depth)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < DEPTH_W && y < DEPTH_H) {
		int id = y * DEPTH_W + x;

		float origin = depth[id];
		float result = 0;
		if (origin != 0) {
			float sum = origin;
			float weight = 1;
			float w = 1;
			for (int r = 1; r <= SF_RADIUS; r++) {
				w *= SF_ALPHA;
				if (y - r >= 0 && depth[id - r * DEPTH_W] != 0 && fabs(depth[id - r * DEPTH_W] - origin) <= SF_THRESHOLD) {
					weight += w;
					sum += w * depth[id - r * DEPTH_W];
				}
				if (y + r < DEPTH_H && depth[id + r * DEPTH_W] != 0 && fabs(depth[id + r * DEPTH_W] - origin) <= SF_THRESHOLD) {
					weight += w;
					sum += w * depth[id + r * DEPTH_W];
				}
			}
			result = sum / weight;
		}
		__syncthreads();
		depth[id] = result;
	}
}

__global__ void kernelSFHorizontal(float* depth)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < DEPTH_W && y < DEPTH_H) {
		int id = y * DEPTH_W + x;

		float origin = depth[id];
		float result = 0;
		if (origin != 0) {
			float sum = origin;
			float weight = 1;
			float w = 1;
			for (int r = 1; r <= SF_RADIUS; r++) {
				w *= SF_ALPHA;
				if (x - r >= 0 && depth[id - r] != 0 && fabs(depth[id - r] - origin) <= SF_THRESHOLD) {
					weight += w;
					sum += w * depth[id - r];
				}
				if (x + r < DEPTH_W && depth[id + r] != 0 && fabs(depth[id + r] - origin) <= SF_THRESHOLD) {
					weight += w;
					sum += w * depth[id + r];
				}
			}
			result = sum / weight;
		}
		__syncthreads();
		depth[id] = result;
	}
}

__global__ void kernelFillHoles(float* depth) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < DEPTH_W && y < DEPTH_H) {
		int id = y * DEPTH_W + x;
		float result = depth[id];
		int cnt = 0;
		if (result == 0) {
			for (int xx = x - 1; xx <= x + 1; xx++) {
				for (int yy = y - 1; yy <= y + 1; yy++) {
					if (0 <= xx && xx < DEPTH_W && 0 <= yy && yy < DEPTH_H && (xx != x || yy != y)) {
						float currDepth = depth[yy * DEPTH_W + xx];
						if (currDepth != 0) {
							cnt++;
							result = max(result, currDepth);
						}
					}
				}
			}
		}
		__syncthreads();
		if (cnt >= 5) {
			depth[id] = result;
		}
	}
}

__global__ void kernelTemporalFilter(float* depth, float* lastFrame) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < DEPTH_W && y < DEPTH_H) {
		int id = y * DEPTH_W + x;
		float result = depth[id];
		float lastDepth = lastFrame[id];
		if (lastDepth != 0 && fabs(result - lastDepth) <= TF_THRESHOLD) {
			result = result * TF_ALPHA + lastDepth * (1 - TF_ALPHA);
		}
		__syncthreads();
		depth[id] = result;
		lastFrame[id] = result;
	}
}

extern "C"
void cudaDepthFilterInit(UINT16*& depth_device, float*& depthFloat_device, float*& lastFrame_device) {
	dim3 threadsPerBlock = dim3(256, 1);
	dim3 blocksPerGrid = dim3((DEPTH_W + threadsPerBlock.x - 1) / threadsPerBlock.x, (DEPTH_H + threadsPerBlock.y - 1) / threadsPerBlock.y);

	HANDLE_ERROR(hipMalloc(&depth_device, DEPTH_H * DEPTH_W * sizeof(UINT16)));
	HANDLE_ERROR(hipMalloc(&depthFloat_device, MAX_CAMERAS * DEPTH_H * DEPTH_W * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&lastFrame_device, MAX_CAMERAS * DEPTH_H * DEPTH_W * sizeof(float)));
	for (int i = 0; i < MAX_CAMERAS; i++) {
		kernelCleanLastFrame << <blocksPerGrid, threadsPerBlock >> > (lastFrame_device + i * DEPTH_H * DEPTH_W);
		hipGetLastError();
	}
}

extern "C"
void cudaDepthFilterClean(UINT16*& depth_device, float*& depthFloat_device, float*& lastFrame_device) {
	HANDLE_ERROR(hipFree(depth_device));
	HANDLE_ERROR(hipFree(depthFloat_device));
	HANDLE_ERROR(hipFree(lastFrame_device));
}

extern "C"
void cudaDepthFiltering(UINT16* depthMap, UINT16* depth_device, float* depthFloat_device, float* lastFrame_device, float convertFactor) {
	dim3 threadsPerBlock = dim3(256, 1);
	dim3 blocksPerGrid = dim3((DEPTH_W + threadsPerBlock.x - 1) / threadsPerBlock.x, (DEPTH_H + threadsPerBlock.y - 1) / threadsPerBlock.y);

	HANDLE_ERROR(hipMemcpy(depth_device, depthMap, DEPTH_H * DEPTH_W * sizeof(UINT16), hipMemcpyHostToDevice));
	kernelFilterToDisparity << <blocksPerGrid, threadsPerBlock >> > (depth_device, depthFloat_device, convertFactor);
	hipGetLastError();

	for (int i = 0; i < 2; i++) {
		kernelSFVertical << <blocksPerGrid, threadsPerBlock >> > (depthFloat_device);
		hipGetLastError();
		kernelSFHorizontal << <blocksPerGrid, threadsPerBlock >> > (depthFloat_device);
		hipGetLastError();
	}

	for (int i = 0; i < 3; i++) {
		kernelFillHoles << <blocksPerGrid, threadsPerBlock >> > (depthFloat_device);
		hipGetLastError();
	}

	kernelTemporalFilter << <blocksPerGrid, threadsPerBlock >> > (depthFloat_device, lastFrame_device);
	hipGetLastError();
	kernelFilterToDepth << <blocksPerGrid, threadsPerBlock >> > (depthFloat_device, convertFactor);
	hipGetLastError();
}
